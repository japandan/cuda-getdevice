#include <stdio.h>

hipDeviceProp_t prop;

int getSPcores(hipDeviceProp_t devProp)
{  
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    switch (devProp.major){
     case 1: // Tesla
      cores = mp * 32;
      break;
     case 2: // Fermi
      if (devProp.minor == 1) cores = mp * 48;
      else cores = mp * 32;
      break;
     case 3: // Kepler
      cores = mp * 192;
      break;
     case 5: // Maxwell
      cores = mp * 128;
      break;
     case 6: // Pascal
      if (devProp.minor == 1) cores = mp * 128;
      else if (devProp.minor == 0) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     case 7: // Volta and Turing
      if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     default:
      printf("Unknown device type\n"); 
      break;
      }
    return cores;
}

main() {
	prop=

}
