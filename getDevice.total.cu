#include <stdio.h>
#include <string>
#include <iostream>
#include <locale.h>   

using namespace std; //no longer require std:: prefix for string functions
 
int getSPcores(hipDeviceProp_t devProp, std::string& arch)
{  
    int cores = 0;
    int mp = devProp.multiProcessorCount;
    arch="unknown";

    switch (devProp.major){
     case 2: // Fermi
      arch="Fermi";
      if (devProp.minor == 1) cores = mp * 48;
      else cores = mp * 32;
      break;
     case 3: // Kepler
      arch="Kepler";
      cores = mp * 192;
      break;
     case 5: // Maxwell
      arch="Maxwell";
      cores = mp * 128;
      break;
     case 6: // Pascal
      arch="Pascal";
      if (devProp.minor == 1) cores = mp * 128;
      else if (devProp.minor == 0) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     case 7: // Volta and Turing
      arch="Volta|Turing";
      if ((devProp.minor == 0) || (devProp.minor == 5)) cores = mp * 64;
      else printf("Unknown device type\n");
      break;
     default:
      printf("Unknown device type\n"); 
      break;
      }
    return cores;
}



int main() {

  setlocale(LC_NUMERIC, ""); //adds comma for big numbers in printf


  string arch("undefined");
  hipDeviceProp_t prop;
  int nDevices;
  int totalCores=0;
  int cudaCores=0;
  hipGetDeviceCount(&nDevices);

  for (int i = 0; i < nDevices; i++) {
    hipGetDeviceProperties(&prop, i);
    printf("Device Number: %d\n", i);
    printf("  Device name: %s\n", prop.name);
    cudaCores=getSPcores(prop, arch);
    printf("  CUDA Cores: %'d\n", cudaCores);
    totalCores=totalCores+cudaCores;
    cout<< "  Architecture: "+arch+"\n";
    printf("  Memory Clock Rate (GHz): %d\n",
           prop.memoryClockRate/1000000);
    printf("  Memory Bus Width (bits): %d\n",
           prop.memoryBusWidth);
    printf("  Peak Memory Bandwidth (GB/s): %f\n\n",
           2.0*prop.memoryClockRate*(prop.memoryBusWidth/8)/1.0e6);


  }

    printf("Total CUDA Cores: %'d\n", totalCores);
}
